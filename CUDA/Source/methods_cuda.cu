#include "hip/hip_runtime.h"
#pragma once

#include "hip/hip_runtime.h"
#include ""

#include <fstream>
#include <iomanip>
#include <iostream>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

// GPU - Init random number generator
__global__ void setupCurandStates(hiprandState* states, unsigned int seed) {
    // hiprandState must be different for each row/thread
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, idx, 0, &states[idx]);
}

// GPU - matrix generation
__global__ void generateMatrixKernel(double* matrix, int n, double B, hiprandState* globalState) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= n) return;

    double dampingValue = (1.0 - B) / n;
    hiprandState localState = globalState[idx];
    // Each row is handled by a separate 
    double* row = &matrix[idx * n];

    // Initialize the row with the damping value
    for (int j = 0; j < n; ++j) {
        row[j] = dampingValue;
    }

    // Generate random edges
    int edges = 1 + hiprand(&localState) % (n - 1);
    double weight = B / edges;

    for (int j = 0; j < edges; ++j) {
        int index = hiprand(&localState) % n;
        row[index] += weight;
    }

    globalState[idx] = localState;
}

// CPU
void generateMatrix(double* matrix, int n, double B) {
    // Allocate memory for random number generator and initialize it
    hiprandState* devStates;
    hipMalloc(&devStates, n * sizeof(hiprandState));

    // Generate seed
    unsigned int seed = time(NULL);
    setupCurandStates << <(n + 255) / 256, 256 >> > (devStates, seed);

    // Generate matrix on GPU
    // We use 256 threads per one block, as I've heard that is ok for NVIDIA GPUs
    generateMatrixKernel << <(n + 255) / 256, 256 >> > (matrix, n, B, devStates);

    hipFree(devStates);
}

// This method is not a part of an algoritm, thus it is not implemented as a parallel method
void printMatrix(double* matrix, int n) {
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            std::cout << matrix[i * n + j] << " ";
        }
        std::cout << std::endl;
    }
}

// This method is not a part of an algoritm, thus it is not implemented as a parallel method
void printVector(double* vector, int n) {
    for (int i = 0; i < n; ++i) {
        std::cout << vector[i] << " ";
    }
    std::cout << std::endl;
}

void saveMatrixToFile(double* matrix, int n, const std::string& directory, const std::string& filename, int precision = 15) {
    std::ofstream file(directory + filename);
    if (file.is_open()) {
        file << std::fixed << std::setprecision(precision);
        for (int i = 0; i < n; ++i) {
            for (int j = 0; j < n; ++j) {
                file << matrix[i * n + j] << " ";
            }
            file << "\n";
        }
        file.close();
    }
    else {
        std::cout << "Cannot open the requested file" << std::endl;
    }
}

void loadMatrixFromFile(double* matrix, int n, const std::string& directory, const std::string& filename) {
    std::ifstream file(directory + filename);
    if (file.is_open()) {
        for (int i = 0; i < n; ++i) {
            for (int j = 0; j < n; ++j) {
                file >> matrix[i * n + j];
            }
        }
        file.close();
    }
    else {
        std::cout << "Cannot open the requested file" << std::endl;
    }
}
