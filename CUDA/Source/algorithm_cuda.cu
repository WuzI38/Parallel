#include "hip/hip_runtime.h"
#pragma once

#include "hip/hip_runtime.h"
#include ""

#include <iostream>

// GPU - multiply vector by matrix, each thread is responsible for multiplying a single row by the vector
__global__ void multiplyVectorByMatrixKernel(double* matrix, double* vector, double* result, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= n) return;

    double sum = 0.0;
    for (int j = 0; j < n; ++j) {
        sum += matrix[j * n + idx] * vector[j];
    }
    result[idx] = sum;
}

// Function to multiply a matrix by a vector
void multiplyVectorByMatrix(double* matrix, double* vector, double* result, int n) {
    multiplyVectorByMatrixKernel << <(n + 255) / 256, 256 >> > (matrix, vector, result, n);
}

// GPU - compare vectors
__global__ void compareVectorsKernel(double* vector1, double* vector2, bool* areEqual, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= n) return;

    double epsilon = 1e-6;
    if (abs(vector1[idx] - vector2[idx]) > epsilon) {
        *areEqual = false;
    }
}

// Function to compare two vectors
bool compareVectors(double* vector1, double* vector2, int n) {
    bool h_areEqual = true;
    bool* d_areEqual;
    hipMalloc(&d_areEqual, sizeof(bool));
    hipMemcpy(d_areEqual, &h_areEqual, sizeof(bool), hipMemcpyHostToDevice);

    compareVectorsKernel << <(n + 255) / 256, 256 >> > (vector1, vector2, d_areEqual, n);

    hipMemcpy(&h_areEqual, d_areEqual, sizeof(bool), hipMemcpyDeviceToHost);
    hipFree(d_areEqual);

    return h_areEqual;
}